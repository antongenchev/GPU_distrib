
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <hiprand/hiprand_kernel.h>
#include <iomanip>
#include <fstream>
#include <chrono>

using namespace std;

// Thread count info
#define NUM_THREADS 128 // Number of threads per block
#define NUM_BLOCKS 16 // Number of blocks in the grid
#define TOTAL_THREADS (NUM_THREADS * NUM_BLOCKS)  // Total number of threads

#define LEN_RESULT 100 // adjust
#define LEN_SHARED_MEM 100 // adjust
#define LEN_InputKernelMainFunc 100 // adjust

// Global variable to hold the ranges
unsigned long long range[2];
unsigned long long threadRanges[TOTAL_THREADS][2]; // To store start and end of range for each thread

__constant__ int shared_memory[LEN_SHARED_MEM];

// Function to parse command-line arguments for range values
void parseCommandLineArguments(int argc, char* argv[]) {
    // Default values
    range[0] = 0;  // range_from default
    range[1] = 0;  // range_to default
    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        // Parse --range_from=
        if (arg.find("--range_from=") == 0) {
            range[0] = std::stoull(arg.substr(13).c_str());
        }
        // Parse --range_to=
        else if (arg.find("--range_to=") == 0) {
            range[1] = std::stoull(arg.substr(11).c_str());
        }
    }
}

void calculateThreadRanges(unsigned long long rangeStart, unsigned long long rangeEnd, unsigned long long threadRanges[TOTAL_THREADS][2]) {
    /* Divide the range into multiple ranges, one for each thread */
    unsigned long long totalRangeSize = rangeEnd - rangeStart + 1;
    unsigned long long chunkSize = totalRangeSize / TOTAL_THREADS; // Size for each thread
    unsigned long long remainder = totalRangeSize % TOTAL_THREADS; // Extra elements to distribute
    unsigned long long currentStart = rangeStart;
    for (int i = 0; i < TOTAL_THREADS; ++i) {
        // Each thread gets at least 'chunkSize' elements, plus one more if there are remainders
        unsigned long long currentEnd = currentStart + chunkSize - 1;
        if (i < remainder) {
            currentEnd += 1; // Distribute the remainder to the first few threads
        }
        // Store the calculated range for this thread
        threadRanges[i][0] = currentStart;
        threadRanges[i][1] = currentEnd;
        // Update the starting point for the next thread
        currentStart = currentEnd + 1;
    }
}

__device__ void getInputKernelMainFunc(unsigned long long n, int* InputKernelMainFunc) {
    /* Given a number n return the corresponding input needed for the KernelMainFunc */
    // TODO
}

__device__ void KernelMainFunc(int InpputKernelMainFunc[], int* result) {
    // Run the main kernel funcion
}

__global__ void CUDA_kernel(unsigned long long* threadRanges, int* result) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= TOTAL_THREADS) return; // Safety check

    int InputKernelMainFunc[LEN_InputKernelMainFunc];

    // Each thread uses its assigned range
    unsigned long long rangeStart = threadRanges[tid * 2];
    unsigned long long rangeEnd = threadRanges[tid * 2 + 1];

    for (unsigned long long n = rangeStart; n <= rangeEnd; ++n) {
        getInputKernelMainFunc(n, InputKernelMainFunc);
        KernelMainFunc(InputKernelMainFunc, result);
    }
}

int main(int argc, char* argv[]) {
    // Parse the command-line arguments for range
    parseCommandLineArguments(argc, argv);

    // Get the range for each thread
    calculateThreadRanges(range[0], range[1], threadRanges);
    unsigned long long flattenedThreadRanges[TOTAL_THREADS * 2];
    for (int i = 0; i < TOTAL_THREADS; ++i) {
        flattenedThreadRanges[i * 2] = threadRanges[i][0];
        flattenedThreadRanges[i * 2 + 1] = threadRanges[i][1];
    }
    // Allocate memory on the GPU for flattened threadRanges
    unsigned long long* d_threadRanges;
    hipMalloc((void**)&d_threadRanges, sizeof(unsigned long long) * 2 * TOTAL_THREADS);
    // Copy flattened threadRanges from host to device
    hipMemcpy(d_threadRanges, flattenedThreadRanges, sizeof(unsigned long long) * 2 * TOTAL_THREADS, hipMemcpyHostToDevice);

    // Allocate memory for storing results
    int* devResults;
	int* hostResults = new int[LEN_RESULT];
    for (int i = 0; i < LEN_RESULT; ++ i) {
        hostResults[i] = 0;
    }
    hipMalloc((void**)&devResults, LEN_RESULT * sizeof(int));

    // Run the CUDA kernel
    auto start = std::chrono::high_resolution_clock::now();
    CUDA_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(d_threadRanges, devResults);
    hipDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = stop - start;
    // Print the time taken by the kernel
    std::cout << "Time taken by kernel: " << duration.count() << " ms" << std::endl;
    // Print any CUDA errors
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    // Copy results back to host from device
    hipMemcpy(hostResults, devResults, LEN_RESULT * sizeof(int), hipMemcpyDeviceToHost);

    // // Write the results to a file:
    std::ofstream outFile("results.txt"); // Create and open file
    // Check if file is opened successfully
    if (outFile.is_open()) {
        // Write results to the file
        for (int i = 0; i < LEN_RESULT; ++i) {
            outFile << "result[" << i << "]: " << hostResults[i] << "\n";
        }
        outFile.close();  // Close the file after writing
        std::cout << "Results successfully written to results.txt\n";
    } else {
        std::cerr << "Error: Could not open file for writing.\n";
    }

    // Free alocated memory on the CPU
    delete[] hostResults;
    // Free alocated memory on the GPU
	hipFree(devResults);

    return 0;
}